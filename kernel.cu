﻿#include "GDI_CUDA_HEADER.cuh"

int main()
{
    Point leftTop = { 0, 0 };
    Point rightBottom = { 200, 200 };
    HDC screenDC = GetDC(NULL);
    BITMAP img0 = { 0 };
    HDC* memDC = NULL;
    HBITMAP* memImg = NULL;
    transScreenDC2Img(screenDC, &img0, leftTop, rightBottom, &memDC, &memImg);
    PIMG img = makeImg(img0, &screenDC, &memDC, &memImg, YES, L"EXEx");
    PRGB_TENSOR rgbTensor = NULL;
    makeImgRgbTensor(&img, &rgbTensor);
    //从现在开始，仅有rgbTensor是可用内存！
    playRgbTensor(&rgbTensor);

    transRgbTensor2ImgOnDisk(&rgbTensor, L"E:\\desk\\woaini.png");
    return 0;
}