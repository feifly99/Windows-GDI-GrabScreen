#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <windows.h>
#include <time.h>

#define IN
#define OUT
#define OUT_PTR
#define SHOW_LOG

#define ASSERT(p) (p) ? 1:exit(0xFFF);

#pragma warning(disable:6387)
#pragma warning(disable:6385)
#pragma warning(disable:6011)
#pragma warning(disable:28183)

typedef struct _Point
{
    int x;
    int y;
}Point, * PPoint;

typedef struct _img
{
    PBITMAPFILEHEADER filePointer;
    PBITMAPINFOHEADER infoPointer;
    PVOID pixelPointer;
}IMG, * PIMG;

typedef struct _pixelRGB
{
    UCHAR R;
    UCHAR G;
    UCHAR B;
}pxlRGB, * ppxlRGB;

pxlRGB getPixelRGB_byRGBPointer(
    IN PVOID firstRGBPointer
)
{
    pxlRGB ret = { 0 };
    ret.R = *(UCHAR*)((ULONG64)firstRGBPointer + 2);
    ret.G = *(UCHAR*)((ULONG64)firstRGBPointer + 1);
    ret.B = *(UCHAR*)((ULONG64)firstRGBPointer + 0);
    return ret;
}

void echo(
    IN SHOW_LOG WCHAR* str
)
{
    MessageBoxW(NULL, str, NULL, MB_OK);
}

void initializeInfoHeader(
    IN BITMAP bmpScreen,
    OUT BITMAPINFOHEADER* bi
)
{
    (*bi).biSize = sizeof(BITMAPINFOHEADER);
    (*bi).biWidth = bmpScreen.bmWidth;
    (*bi).biHeight = bmpScreen.bmHeight;
    (*bi).biPlanes = 1;
    (*bi).biBitCount = 32;
    (*bi).biCompression = BI_RGB;
    (*bi).biSizeImage = 0;
    (*bi).biXPelsPerMeter = 0;
    (*bi).biYPelsPerMeter = 0;
    (*bi).biClrUsed = 0;
    (*bi).biClrImportant = 0;
    return;
}

void initializeFileHeader(
    OUT PBITMAPFILEHEADER imgFileHeader,
    IN DWORD sizeHeaderAndInfo,
    IN DWORD wholeSize,
    IN WORD imgHeadMarkCode
)
{
    imgFileHeader->bfOffBits = sizeHeaderAndInfo;
    imgFileHeader->bfSize = wholeSize;
    imgFileHeader->bfType = 0x4D42;
    return;
}

void initializePixelHeader(
    OUT_PTR PVOID* imgPixelHeader,
    IN LONG imgPixelSize,
    IN HDC nakedScreenImage,
    IN HBITMAP hiddenImageInstance,
    IN BITMAP img,
    IN BITMAPINFO* imgInfoHeader
)
{
    *imgPixelHeader = malloc((SIZE_T)imgPixelSize);
    GetDIBits(
        nakedScreenImage,
        hiddenImageInstance,
        0,
        (UINT)img.bmHeight,
        *imgPixelHeader,
        imgInfoHeader,
        DIB_RGB_COLORS
    );
}

void createImageFile(
    IN WCHAR* w_filePath,
    OUT HANDLE* hFile
)
{
    *hFile = CreateFileW(
        w_filePath,
        GENERIC_WRITE,
        0,
        NULL,
        CREATE_ALWAYS,
        FILE_ATTRIBUTE_NORMAL,
        NULL
    );
}

SIZE_T getTotalBitMapSize(
    IN DWORD dwBmpSize
)
{
    return (SIZE_T)dwBmpSize + sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER);
}

LONG getBmpSize(
    IN BITMAP bmpScreen,
    IN BITMAPINFOHEADER bi
)
{
    return ((bmpScreen.bmWidth * bi.biBitCount + 31) / 32) * 4 * bmpScreen.bmHeight;
}

void saveImg2Disk(
    OUT HANDLE* hFile,
    CONST WCHAR* fatherLocWithNoMark,
    CONST WCHAR* imgName,
    IN PBITMAPFILEHEADER imgFileHeader,
    IN PBITMAPINFOHEADER imgInfoHeader,
    IN PVOID imgPixelHeader,
    IN DWORD wholeSize
)
{
    WCHAR FilePath[MAX_PATH];
    wsprintfW(FilePath, L"%s\\%s.png", fatherLocWithNoMark, imgName);

    createImageFile(FilePath, hFile);
    WriteFile(*hFile, (LPSTR)imgFileHeader, sizeof(BITMAPFILEHEADER), NULL, NULL);
    WriteFile(*hFile, (LPSTR)imgInfoHeader, sizeof(BITMAPINFOHEADER), NULL, NULL);
    WriteFile(*hFile, (LPSTR)imgPixelHeader, wholeSize, NULL, NULL);
    return;
}

void transScreenDC2Img(
    IN HDC screenDC,
    OUT BITMAP* img,
    IN Point lt,
    IN Point rb,
    OUT_PTR HDC** _hiddenImgCode,
    OUT_PTR HBITMAP** _hiddenImageInstance
)
{
    int w = rb.x - lt.x;
    int h = rb.y - lt.y;
    *_hiddenImgCode = (HDC*)malloc(sizeof(HDC));
    *_hiddenImageInstance = (HBITMAP*)malloc(sizeof(HBITMAP));
    **_hiddenImgCode = CreateCompatibleDC(screenDC);
    **_hiddenImageInstance = CreateCompatibleBitmap(screenDC, w, h);

    SelectObject(**_hiddenImgCode, **_hiddenImageInstance);
    BitBlt(
        **_hiddenImgCode,
        0, 0,
        w, h,
        screenDC,
        lt.x, lt.y,
        SRCCOPY
    );
    GetObjectW(**_hiddenImageInstance, sizeof(BITMAP), img);
    return;
}

PIMG makeImg(
    IN BITMAP img,
    IN HDC nakedScreenImage,
    IN HBITMAP memImg,
    IN BOOLEAN isSaveToDisk
)
{
    PIMG ret = (PIMG)malloc(sizeof(IMG));

    PBITMAPINFOHEADER imgInfoHeaderPointer = (PBITMAPINFOHEADER)malloc(sizeof(BITMAPINFOHEADER));
    PBITMAPFILEHEADER imgFileHeaderPointer = (PBITMAPFILEHEADER)malloc(sizeof(BITMAPFILEHEADER));
    PVOID imgPixelHeader = NULL;

    initializeInfoHeader(img, imgInfoHeaderPointer);

    LONG imgPixelSize = getBmpSize(img, *imgInfoHeaderPointer);
    initializePixelHeader(
        &imgPixelHeader,
        imgPixelSize,
        nakedScreenImage,
        memImg,
        img,
        (BITMAPINFO*)imgInfoHeaderPointer
    );

    initializeFileHeader(
        imgFileHeaderPointer,
        (DWORD)sizeof(BITMAPFILEHEADER) + (DWORD)sizeof(BITMAPINFOHEADER),
        imgPixelSize + sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER),
        0x4D42
    );

    if (isSaveToDisk)
    {
        HANDLE hFile = NULL;
        saveImg2Disk(
            &hFile,
            L"E:\\desk",
            L"newScreenSnapShot",
            imgFileHeaderPointer,
            imgInfoHeaderPointer,
            imgPixelHeader,
            imgPixelSize + sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER)
        );
        CloseHandle(hFile);
    }

    ret->filePointer = imgFileHeaderPointer;
    ret->infoPointer = imgInfoHeaderPointer;
    ret->pixelPointer = imgPixelHeader;

    return ret;
}

void ExFreeMem(
    PVOID* mem
)
{
    if (*mem == NULL)
    {
        printf("空指针传入.\n");
        return;
    }
    free(*mem);
    *mem = NULL;
    return;
}

int main()
{
    Point leftTop = { 0, 0 };
    Point rightBottom = { 1920, 1080 };

    HDC screenDC = GetDC(NULL);
    BITMAP img0 = { 0 };
    HDC* memDC = NULL;
    HBITMAP* memImg = NULL;
    transScreenDC2Img(screenDC, &img0, leftTop, rightBottom, &memDC, &memImg);
    PIMG img = makeImg(img0, screenDC, *memImg, 1);
    ExFreeMem((PVOID*)&img->filePointer);
    ExFreeMem((PVOID*)&img->infoPointer);
    ExFreeMem((PVOID*)&img->pixelPointer);
    DeleteObject(*memImg);
    DeleteObject(*memDC);
    ExFreeMem((PVOID*)&memImg);
    ExFreeMem((PVOID*)&memDC);
    ReleaseDC(NULL, screenDC);
    ExFreeMem((PVOID*)&img);

    return 0;
}
