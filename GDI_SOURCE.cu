#include "hip/hip_runtime.h"
#include "GDI_CUDA_HEADER.cuh"

void ExFreeMem(
    PVOID* mem
)
{
    if (*mem == NULL)
    {
        printf("��ָ�봫��.\n");
        return;
    }
    free(*mem);
    *mem = NULL;
    return;
}

pxlRGB getPixelRGB_byRGBPointer(
    IN PVOID firstRGBPointer
)
{
    pxlRGB ret = { 0 };
    ret.R = *(UCHAR*)((ULONG64)firstRGBPointer + 2);
    ret.G = *(UCHAR*)((ULONG64)firstRGBPointer + 1);
    ret.B = *(UCHAR*)((ULONG64)firstRGBPointer + 0);
    return ret;
}

void echo(
    IN SHOW_LOG WCHAR* str
)
{
    MessageBoxW(NULL, str, NULL, MB_OK);
}

void initializeInfoHeader(
    IN BITMAP bmpScreen,
    OUT BITMAPINFOHEADER* bi
)
{
    (*bi).biSize = sizeof(BITMAPINFOHEADER);
    (*bi).biWidth = bmpScreen.bmWidth;
    (*bi).biHeight = bmpScreen.bmHeight;
    (*bi).biPlanes = 1;
    (*bi).biBitCount = 32;
    (*bi).biCompression = BI_RGB;
    (*bi).biSizeImage = 0;
    (*bi).biXPelsPerMeter = 0;
    (*bi).biYPelsPerMeter = 0;
    (*bi).biClrUsed = 0;
    (*bi).biClrImportant = 0;
    return;
}

void initializeFileHeader(
    OUT PBITMAPFILEHEADER imgFileHeader,
    IN DWORD sizeHeaderAndInfo,
    IN DWORD wholeSize,
    IN WORD imgHeadMarkCode
)
{
    imgFileHeader->bfOffBits = sizeHeaderAndInfo;
    imgFileHeader->bfSize = wholeSize;
    imgFileHeader->bfType = 0x4D42;
    return;
}

void initializePixelHeader(
    OUT_PTR PVOID* imgPixelHeader,
    IN LONG imgPixelSize,
    IN HDC nakedScreenImage,
    IN HBITMAP hiddenImageInstance,
    IN BITMAP img,
    IN BITMAPINFO* imgInfoHeader
)
{
    *imgPixelHeader = malloc((SIZE_T)imgPixelSize);
    GetDIBits(
        nakedScreenImage,
        hiddenImageInstance,
        0,
        (UINT)img.bmHeight,
        *imgPixelHeader,
        imgInfoHeader,
        DIB_RGB_COLORS
    );
}

void createImageFile(
    IN WCHAR* w_filePath,
    OUT HANDLE* hFile
)
{
    *hFile = CreateFileW(
        w_filePath,
        GENERIC_WRITE,
        0,
        NULL,
        CREATE_ALWAYS,
        FILE_ATTRIBUTE_NORMAL,
        NULL
    );
}

SIZE_T getTotalBitMapSize(
    IN DWORD dwBmpSize
)
{
    return (SIZE_T)dwBmpSize + sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER);
}

LONG getBmpSize(
    IN BITMAP bmpScreen,
    IN BITMAPINFOHEADER bi
)
{
    return ((bmpScreen.bmWidth * bi.biBitCount + 31) / 32) * 4 * bmpScreen.bmHeight;
}

void saveImg2Disk(
    OUT HANDLE* hFile,
    CONST WCHAR* fatherLocWithNoMark,
    CONST WCHAR* imgName,
    IN PBITMAPFILEHEADER imgFileHeader,
    IN PBITMAPINFOHEADER imgInfoHeader,
    IN PVOID imgPixelHeader,
    IN DWORD wholeSize
)
{
    WCHAR FilePath[MAX_PATH];
    wsprintfW(FilePath, L"%s\\%s.png", fatherLocWithNoMark, imgName);

    createImageFile(FilePath, hFile);
    WriteFile(*hFile, (LPSTR)imgFileHeader, sizeof(BITMAPFILEHEADER), NULL, NULL);
    WriteFile(*hFile, (LPSTR)imgInfoHeader, sizeof(BITMAPINFOHEADER), NULL, NULL);
    WriteFile(*hFile, (LPSTR)imgPixelHeader, wholeSize, NULL, NULL);
    return;
}

void transScreenDC2Img(
    IN HDC screenDC,
    OUT BITMAP* img,
    IN Point lt,
    IN Point rb,
    OUT_PTR HDC** _hiddenImgCode,
    OUT_PTR HBITMAP** _hiddenImageInstance
)
{
    int w = rb.x - lt.x;
    int h = rb.y - lt.y;
    *_hiddenImgCode = (HDC*)malloc(sizeof(HDC));
    *_hiddenImageInstance = (HBITMAP*)malloc(sizeof(HBITMAP));
    **_hiddenImgCode = CreateCompatibleDC(screenDC);
    **_hiddenImageInstance = CreateCompatibleBitmap(screenDC, w, h);

    SelectObject(**_hiddenImgCode, **_hiddenImageInstance);
    BitBlt(
        **_hiddenImgCode,
        0, 0,
        w, h,
        screenDC,
        lt.x, lt.y,
        SRCCOPY
    );
    GetObjectW(**_hiddenImageInstance, sizeof(BITMAP), img);
    return;
}

PIMG makeImg(
    IN BITMAP img,
    IN_CLEAR_OUT HDC* screenDC,
    IN_CLEAR_OUT HDC** memDC,
    IN_CLEAR_OUT HBITMAP** memImg,
    IN BOOLEAN isSaveToDisk,
    IN_OPT CONST WCHAR* name
)
{
    PIMG ret = (PIMG)malloc(sizeof(IMG));

    PBITMAPINFOHEADER imgInfoHeaderPointer = (PBITMAPINFOHEADER)malloc(sizeof(BITMAPINFOHEADER));
    PBITMAPFILEHEADER imgFileHeaderPointer = (PBITMAPFILEHEADER)malloc(sizeof(BITMAPFILEHEADER));
    PVOID imgPixelHeader = NULL;

    initializeInfoHeader(img, imgInfoHeaderPointer);

    LONG imgPixelSize = getBmpSize(img, *imgInfoHeaderPointer);
    initializePixelHeader(
        &imgPixelHeader,
        imgPixelSize,
        *screenDC,
        **memImg,
        img,
        (BITMAPINFO*)imgInfoHeaderPointer
    );

    initializeFileHeader(
        imgFileHeaderPointer,
        (DWORD)sizeof(BITMAPFILEHEADER) + (DWORD)sizeof(BITMAPINFOHEADER),
        imgPixelSize + sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER),
        0x4D42
    );

    if (isSaveToDisk && name)
    {
        HANDLE hFile = NULL;
        saveImg2Disk(
            &hFile,
            L"E:\\desk",
            name,
            imgFileHeaderPointer,
            imgInfoHeaderPointer,
            imgPixelHeader,
            imgPixelSize + sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER)
        );
        CloseHandle(hFile);
    }
    else
    {
        printf("isSaveToDisk && name == 0, �������.\n");
        exit(0xFFFFFFCF);
    }

    ret->filePointer = imgFileHeaderPointer;
    ret->infoPointer = imgInfoHeaderPointer;
    ret->pixelPointer = imgPixelHeader;

    DeleteObject(**memImg);
    ExFreeMem((PVOID*)&(*memImg));
    DeleteObject(**memDC);
    ExFreeMem((PVOID*)&(*memDC));
    ReleaseDC(NULL, *screenDC);
    return ret;
}

void makeImgRgbTensor(
    IN_CLEAR_OUT PIMG* img,
    PRGB_TENSOR* tensor
)
{
    if (*tensor != NULL)
    {
        printf("�Ѿ���ֵ.\n");
        exit(0xEEFFEE);
    }
    *tensor = (PRGB_TENSOR)malloc(sizeof(RGB_TENSOR));
    (*tensor)->width = (*img)->infoPointer->biWidth;
    (*tensor)->height = (*img)->infoPointer->biHeight;
    (*tensor)->depth = 3;
    (*tensor)->RGBtensor = (UCHAR***)malloc((*tensor)->depth * sizeof(UCHAR**));
    fors(
        (*tensor)->depth,
        ((*tensor)->RGBtensor)[j] = (UCHAR**)malloc((*tensor)->height * sizeof(UCHAR*));
    );
    forss(
        (*tensor)->depth, (*tensor)->height,
        ((*tensor)->RGBtensor)[j][i] = (UCHAR*)malloc((*tensor)->width * sizeof(UCHAR));,
    );
    ULONG64 headerPixelPointer = (ULONG64)((*img)->pixelPointer) + (*tensor)->width * (*tensor)->height * ((*tensor)->depth + 1) - 4;
    forss(
		(*tensor)->height, (*tensor)->width,
        ((*tensor)->RGBtensor)[0][j][(*tensor)->width - i - 1] = *(UCHAR*)(headerPixelPointer + 2); //R
		((*tensor)->RGBtensor)[1][j][(*tensor)->width - i - 1] = *(UCHAR*)(headerPixelPointer + 1); //G
		((*tensor)->RGBtensor)[2][j][(*tensor)->width - i - 1] = *(UCHAR*)(headerPixelPointer + 0); //B
		headerPixelPointer -= 4; ,
    );
    ExFreeMem((PVOID*)&((*img)->filePointer));
    ExFreeMem((PVOID*)&((*img)->infoPointer));
    ExFreeMem((PVOID*)&((*img)->pixelPointer));
    ExFreeMem((PVOID*)&((*img)));
    return;
}

static void makeCudaFriendlyDimensionAligned(
	IN_OUT SIZE_T (*dimensions)[MAX_DIMENSION_SIZE],
	IN SIZE_T alignment
)
{
	if ((*dimensions) == NULL)
	{
		printf("���Ϸ���ά�����룬�Ѳ���.\n");
		exit(0xA000);
	}

	fors(
		MAX_DIMENSION_SIZE,
		(*dimensions)[j] = ((*dimensions)[j] + alignment - 1) & ~(alignment - 1);
	);
	return;
}

OUT_HOST_PTR PDATA_SET makeDataSet(
	IN myCudaDataFlag inputDataFlag,
	IN DATA_GENERATE_TYPE dataGenerateType,
	IN_OUT SIZE_T(*dimensions)[MAX_DIMENSION_SIZE],
	IN SIZE_T alignmentValue,
	IN_OPT PVOID existData
)
{
	SIZE_T dimension0 = (*dimensions)[0];
	SIZE_T dimension1 = (*dimensions)[1];
	SIZE_T dimension2 = (*dimensions)[2];
	makeCudaFriendlyDimensionAligned(dimensions, alignmentValue);
	PDATA_SET ret = (PDATA_SET)malloc(sizeof(DATA_SET));
	ret->dataFlag = inputDataFlag;
	if (ret->dataFlag == CUDA_DATA_FLAG_$_VECTOR_$)
	{
		if (dimension1 != 0 || dimension2 != 0)
		{
			printf("����ά�����������Ϳ��ܲ������Ѳ���.\n");
			free(ret);
			ret = NULL;
			exit(0xFA00);
		}

		ret->dataType.Vector.size = (*dimensions)[0];

		ret->dataType.Vector.vector = (float*)malloc((*dimensions)[0] * sizeof(float));
		RtlZeroMemory(ret->dataType.Vector.vector, (*dimensions)[0]);

		ret->dataType.Vector.objectReference = 1;

		if (dataGenerateType == GENERATE_RANDOM)
		{
			fors(
				ret->dataType.Vector.size,
				ret->dataType.Vector.vector[j] = (float)(rand() % 20 + 1);
				);
		}
		else if (dataGenerateType == GENERATE_ALL_ZERO)
		{
			fors(
				ret->dataType.Vector.size,
				ret->dataType.Vector.vector[j] = 0.0f;
				);
		}
		else if (dataGenerateType == GENERATE_ALL_ONE)
		{
			fors(
				ret->dataType.Vector.size,
				ret->dataType.Vector.vector[j] = 1.0f;
				);
		}
		else if (dataGenerateType == GENERATE_TEST)
		{
			fors(
				ret->dataType.Vector.size,
				ret->dataType.Vector.vector[j] = (float)(j);
				);
		}
		else if (dataGenerateType == GENERATE_EXISTING)
		{
			if (existData == NULL)
			{
				printf("ѡ�����Ѵ������ݽ��и��Ƶ�������ָ��Ϊ�գ��Ѳ���.\n");
				free(ret->dataType.Vector.vector);
				ret->dataType.Vector.vector = NULL;
				free(ret);
				ret = NULL;
				exit(0xFAC0);
			}
			else
			{
				if (dimension0 <= (*dimensions)[0])
				{
					fors(
						ret->dataType.Vector.size,
						ret->dataType.Vector.vector[j] = ((float*)existData)[j];
					);
				}
			}
		}
		else
		{
			printf("�������Ͳ������Ѳ���.\n");
			free(ret->dataType.Vector.vector);
			ret->dataType.Vector.vector = NULL;
			free(ret);
			ret = NULL;
			exit(0xFAF0);
		}
	}
	else if (ret->dataFlag == CUDA_DATA_FLAG_$_MATRIX_$)
	{
		if (dimension2 != 0)
		{
			printf("����ά�����������Ͳ������Ѳ���.\n");
			free(ret);
			ret = NULL;
			exit(0xFA00);
		}

		ret->dataType.Matrix.rowSize = (*dimensions)[0];
		ret->dataType.Matrix.colSize = (*dimensions)[1];

		ret->dataType.Matrix.matrix = (float**)malloc((*dimensions)[0] * sizeof(float*));

		fors(
			(*dimensions)[0],
			ret->dataType.Matrix.matrix[j] = (float*)malloc((*dimensions)[1] * sizeof(float));
		);
		forss(
			(*dimensions)[0], (*dimensions)[1],
			ret->dataType.Matrix.matrix[j][i] = 0.0f; ,
			);

		ret->dataType.Matrix.objectReference = 1;

		if (dataGenerateType == GENERATE_RANDOM)
		{
			forss(
				(*dimensions)[0], (*dimensions)[1],
				ret->dataType.Matrix.matrix[j][i] = (float)(rand() % 20 + 1); ,
				);
		}
		else if (dataGenerateType == GENERATE_ALL_ZERO)
		{
			forss(
				(*dimensions)[0], (*dimensions)[1],
				ret->dataType.Matrix.matrix[j][i] = 0.0f; ,
				);
		}
		else if (dataGenerateType == GENERATE_ALL_ONE)
		{
			forss(
				(*dimensions)[0], (*dimensions)[1],
				ret->dataType.Matrix.matrix[j][i] = 1.0f; ,
				);
		}
		else if (dataGenerateType == GENERATE_TEST)
		{
			forss(
				(*dimensions)[0], (*dimensions)[1],
				ret->dataType.Matrix.matrix[j][i] = (float)((float)(j + 1) / (float)20.0); ,
				);
		}
		else if (dataGenerateType == GENERATE_EXISTING)
		{
			if (existData == NULL)
			{
				printf("ѡ�����Ѵ������ݽ��и��Ƶ�������ָ��Ϊ�գ��Ѳ���.\n");

				fors(
					(*dimensions)[0],
					free(ret->dataType.Matrix.matrix[j]);
				ret->dataType.Matrix.matrix[j] = NULL;
					);
				free(ret->dataType.Matrix.matrix);
				ret->dataType.Matrix.matrix = NULL;
				free(ret);
				ret = NULL;
				exit(0xFAC0);
			}
			else
			{
				if (dimension0 <= (*dimensions)[0] && dimension1 <= (*dimensions)[1])
				{
					forss(
						dimension0, dimension1,
						ret->dataType.Matrix.matrix[j][i] = (float)(((UCHAR**)existData)[j][i]); ,
					);
				}
			}
		}
		else
		{
			printf("�������Ͳ������Ѳ���.\n");
			fors(
				(*dimensions)[0],
				free(ret->dataType.Matrix.matrix[j]);
			ret->dataType.Matrix.matrix[j] = NULL;
				);
			free(ret->dataType.Matrix.matrix);
			ret->dataType.Matrix.matrix = NULL;
			exit(0xFAF0);
		}
	}
	else
	{
		printf("�������Ͳ������Ѳ���.\n");
		free(ret);
		ret = NULL;
		exit(0xC000);
	}
	return ret;
}

void checkDataSet(
	IN PDATA_SET dataSet
)
{
	if (dataSet->dataFlag == 1)
	{
		//vector
		if (!dataSet->dataType.Vector.objectReference)
		{
			printf("�������ü���Ϊ�㣬����Ϊ�գ��Ѳ���.\n");
		}
		printf("���ݼ�: ����\n");
		printf("��������: %zu\n", dataSet->dataType.Vector.size);
		printf("��ǰ���ü���: %zu\n", dataSet->dataType.Vector.objectReference);
		printf("������ֵ: \n");
		fors(
			dataSet->dataType.Vector.size,
			printf("%.2f\t", dataSet->dataType.Vector.vector[j]);
		);
		QAQ;
		return;
	}
	else if (dataSet->dataFlag == 2)
	{
		//matrix
		if (!dataSet->dataType.Matrix.objectReference)
		{
			printf("�������ü���Ϊ�㣬����Ϊ�գ��Ѳ���.\n");
		}
		printf("���ݼ�: ����\n");
		printf("��������: %zu\n", dataSet->dataType.Matrix.rowSize);
		printf("��������: %zu\n", dataSet->dataType.Matrix.colSize);
		printf("��ǰ���ü���: %zu\n", dataSet->dataType.Matrix.objectReference);
		printf("������ֵ: \n");
		forss(
			dataSet->dataType.Matrix.rowSize, dataSet->dataType.Matrix.colSize,
			printf("%.1f\t", dataSet->dataType.Matrix.matrix[j][i]); ,
			QAQ;
		);
	}
	else
	{
		printf("�Ƿ��������ͣ��Ѳ���.\n");
	}
	return;
}

void makeCudaFriendlyData(
	IN PDATA_SET hostDataSet,
	OUT_HOST_PTR float** deviceDataSet
)
{
	if (hostDataSet->dataFlag == CUDA_DATA_FLAG_$_VECTOR_$)
	{
		//vector
		if (!hostDataSet->dataType.Vector.objectReference)
		{
			printf("�������ü���Ϊ�㣬����Ϊ�գ��Ѳ���.\n");
		}
		*deviceDataSet = (float*)malloc(hostDataSet->dataType.Vector.size * sizeof(float));
		memcpy(*deviceDataSet, hostDataSet->dataType.Vector.vector, hostDataSet->dataType.Vector.size * sizeof(float));
		return;
	}
	else if (hostDataSet->dataFlag == CUDA_DATA_FLAG_$_MATRIX_$)
	{
		//matrix
		if (!hostDataSet->dataType.Matrix.objectReference)
		{
			printf("�������ü���Ϊ�㣬����Ϊ�գ��Ѳ���.\n");
		}
		*deviceDataSet = (float*)malloc(hostDataSet->dataType.Matrix.rowSize * hostDataSet->dataType.Matrix.colSize * sizeof(float));
		for (size_t j = 0; j < hostDataSet->dataType.Matrix.rowSize; j++)
		{
			for (size_t i = 0; i < hostDataSet->dataType.Matrix.colSize; i++)
			{
				(*deviceDataSet)[i + j * hostDataSet->dataType.Matrix.colSize] = hostDataSet->dataType.Matrix.matrix[j][i];
			}
		}
	}
	else
	{
		printf("�Ƿ��������ͣ��Ѳ���.\n");
	}
	return;
}

void safetyCheck(
	IN PDATA_SET dataSet1,
	IN PDATA_SET dataSet2,
	IN myCudaDataFlag dataFlag,
	IN myCudaCalculateFlag calculateFlag
)
{
	if (dataSet1->dataFlag != dataFlag || dataSet2->dataFlag != dataFlag)
	{
		printf("�������ݼ������ͺ͵��õ����ݼ����Ͳ�ͬ���Ѳ���.\n");
		exit(0xECC0);
	}
	if (dataFlag == CUDA_DATA_FLAG_$_VECTOR_$)
	{
		if (calculateFlag == CUDA_CAU_TYPE_$_ADD_$)
		{
			if (dataSet1->dataType.Vector.size != dataSet2->dataType.Vector.size)
			{
				printf("���������ĳߴ粻ͬ���Ѳ���.\n");
				exit(0xDCC0);
			}
		}
		else if (calculateFlag == CUDA_CAU_TYPE_$_MULTIPLE_$)
		{
			if (dataSet1->dataType.Vector.size != dataSet2->dataType.Vector.size)
			{
				printf("���������ĳߴ粻ͬ���Ѳ���.\n");
				exit(0xDCC0);
			}
		}
		else
		{
			printf("���õļ������Ͳ���֧�֣��Ѳ���.\n");
			exit(0xDCC0);
		}
	}
	else if (dataFlag == CUDA_DATA_FLAG_$_MATRIX_$)
	{
		if (calculateFlag == CUDA_CAU_TYPE_$_ADD_$)
		{
			if
				(
					(dataSet1->dataType.Matrix.rowSize != dataSet2->dataType.Matrix.rowSize)
					||
					(dataSet2->dataType.Matrix.colSize != dataSet2->dataType.Matrix.colSize)
					)
			{
				printf("��ͼִ�о���ӷ�������������������в�����ͬ���Ѳ���.\n");
				exit(0xDCC8);
			}
		}
		else if (calculateFlag == CUDA_CAU_TYPE_$_MULTIPLE_$)
		{
			if (dataSet1->dataType.Matrix.colSize != dataSet2->dataType.Matrix.rowSize)
			{
				printf("��ͼִ�о���˷������ǵ�һ��������к͵ڶ���������в�����ͬ���Ѳ���.\n");
				exit(0xDCC8);
			}
		}
		else
		{
			printf("���õļ������Ͳ���֧�֣��Ѳ���.\n");
			exit(0xDCC0);
		}
	}
	else
	{
		printf("������������ʹ����Ѳ���.\n");
		exit(0xACC0);
	}
}

static void ExFreeDataSet_$_VECTOR_$(
	IN_OUT PDATA_SET* dataSet
)
{
	if ((*dataSet)->dataFlag != CUDA_DATA_FLAG_$_VECTOR_$)
	{
		printf("�������Ͳ������Ѳ���.\n");
		exit(0xCFF0);
	}
	ExFreeMem((PVOID*)&((*dataSet)->dataType.Vector.vector));
	ExFreeMem((PVOID*)&(*dataSet));
	return;
}

static void ExFreeDataSet_$_MATRIX_$(
	IN_OUT PDATA_SET* dataSet
)
{
	if ((*dataSet)->dataFlag != CUDA_DATA_FLAG_$_MATRIX_$)
	{
		printf("�������Ͳ������Ѳ���.\n");
		exit(0xCFF0);
	}
	fors(
		(*dataSet)->dataType.Matrix.rowSize,
		ExFreeMem((PVOID*)&((*dataSet)->dataType.Matrix.matrix[j]));
	);
	ExFreeMem((PVOID*)&((*dataSet)->dataType.Matrix.matrix));
	ExFreeMem((PVOID*)&(*dataSet));
	return;
}

void ExFreeDataSet(
	IN_OUT PDATA_SET* dataSet
)
{
	switch ((*dataSet)->dataFlag)
	{
	case CUDA_DATA_FLAG_$_VECTOR_$:
		ExFreeDataSet_$_VECTOR_$(dataSet);
		break;
	case CUDA_DATA_FLAG_$_MATRIX_$:
		ExFreeDataSet_$_MATRIX_$(dataSet);
		break;
	default:
		printf("�Ƿ��������ͣ��Ѳ���.\n");
		break;
	}
	return;
}

__global__ void myCudaVectorAdd()
{

	return;
}

__global__ void myCudaVectorMultiple()
{

	return;
}

__global__ void myCudaMatrixAdd()
{

	return;
}

__global__ void myCudaMatrixMultiple(
	float* data1,
	float* data2,
	size_t row1,
	size_t col1,
	size_t row2,
	size_t col2,
	float* ret
)
{
	UINT currRow = blockIdx.x * blockDim.x + threadIdx.x;
	UINT currCol = blockIdx.y * blockDim.y + threadIdx.y;
	float sum = 0.0;
	for (size_t j = 0; j < col1; j++)
	{
		sum += data1[currRow * col1 + j] * data2[j * col2 + currCol] + currRow * 10.0;//weightAngle;
	}
	ret[currRow * col2 + currCol] = sum;
	return;
}

void cudaExFreeMem(
	IN_OUT PVOID* _GPU_mem
)
{
	hipFree(*_GPU_mem);
	*_GPU_mem = NULL;
	return;
}

void callCuda(
	IN PDATA_SET dataSet1,
	IN PDATA_SET dataSet2,
	IN myCudaDataFlag dataFlag,
	IN myCudaCalculateFlag calculateFlag,
	OUT_HOST_PTR float** ret
)
{
	safetyCheck(dataSet1, dataSet2, dataFlag, calculateFlag);

	float* _host_linerData1 = NULL;
	float* _host_linerData2 = NULL;

	makeCudaFriendlyData(dataSet1, &_host_linerData1);
	makeCudaFriendlyData(dataSet2, &_host_linerData2);

	float* _device_linerData1 = NULL;
	float* _device_linerData2 = NULL;

	float* _device_result = NULL;
	size_t _public_resultSize = 0x0;

	if (dataFlag == CUDA_DATA_FLAG_$_VECTOR_$ && calculateFlag == CUDA_CAU_TYPE_$_ADD_$)
	{
		_public_resultSize = dataSet1->dataType.Vector.size;
		hipMalloc(&_device_result, _public_resultSize * sizeof(float));
	}
	if (dataFlag == CUDA_DATA_FLAG_$_VECTOR_$ && calculateFlag == CUDA_CAU_TYPE_$_MULTIPLE_$)
	{
		_public_resultSize = 1;
		hipMalloc(&_device_result, _public_resultSize * sizeof(float));
	}
	if (dataFlag == CUDA_DATA_FLAG_$_MATRIX_$ && calculateFlag == CUDA_CAU_TYPE_$_ADD_$)
	{
		_public_resultSize = dataSet1->dataType.Matrix.rowSize * dataSet1->dataType.Matrix.colSize;
		hipMalloc(&_device_result, _public_resultSize * sizeof(float));
	}
	if (dataFlag == CUDA_DATA_FLAG_$_MATRIX_$ && calculateFlag == CUDA_CAU_TYPE_$_MULTIPLE_$)
	{
		_public_resultSize = dataSet1->dataType.Matrix.rowSize * dataSet2->dataType.Matrix.colSize;
		printf("kernel: _public_resultSize: %zu Bytes CUDA Memory Allocated.\n", _public_resultSize);
		hipMalloc(&_device_result, _public_resultSize * sizeof(float));

		hipMalloc(&_device_linerData1, dataSet1->dataType.Matrix.rowSize * dataSet1->dataType.Matrix.colSize * sizeof(float));
		hipMalloc(&_device_linerData2, dataSet2->dataType.Matrix.rowSize * dataSet2->dataType.Matrix.colSize * sizeof(float));

		hipMemcpy(_device_linerData1, _host_linerData1, dataSet1->dataType.Matrix.rowSize * dataSet1->dataType.Matrix.colSize * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(_device_linerData2, _host_linerData2, dataSet2->dataType.Matrix.rowSize * dataSet2->dataType.Matrix.colSize * sizeof(float), hipMemcpyHostToDevice);

		dim3 threadEx = { 0 };
		threadEx.x = 32;
		threadEx.y = 32;
		threadEx.z = 1;

		dim3 blockEx = { 0 };
		blockEx.x = 100;
		blockEx.y = 2;
		blockEx.z = 1;

		size_t data1_row = dataSet1->dataType.Matrix.rowSize;
		size_t data1_col = dataSet1->dataType.Matrix.colSize;
		size_t data2_row = dataSet2->dataType.Matrix.rowSize;
		size_t data2_col = dataSet2->dataType.Matrix.colSize;

		hipEvent_t start, stop;
		float elapsedTime;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);

		myCudaMatrixMultiple << <blockEx, threadEx >> > (_device_linerData1, _device_linerData2, data1_row, data1_col, data2_row, data2_col, _device_result);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsedTime, start, stop);

		printf("CUDA time: %f ms\n", elapsedTime);
	}
	//��������CUDA��ʽ��������Ϊ������ֵ������С��ǰ�ĵ�resultSize��ͬ��
	float* _host_result = NULL;
	_host_result = (float*)malloc(_public_resultSize * sizeof(float));
	if (_host_result != NULL)
	{
		hipMemcpy(_host_result, _device_result, _public_resultSize * sizeof(float), hipMemcpyDeviceToHost);
		*ret = _host_result;

		cudaExFreeMem((PVOID*)&_device_result);
		cudaExFreeMem((PVOID*)&_device_linerData2);
		cudaExFreeMem((PVOID*)&_device_linerData1);
		ExFreeMem((PVOID*)&_host_linerData2);
		ExFreeMem((PVOID*)&_host_linerData1);
	}
	else
	{
		ExFreeMem((PVOID*)&_host_result);

		cudaExFreeMem((PVOID*)&_device_result);
		cudaExFreeMem((PVOID*)&_device_linerData2);
		cudaExFreeMem((PVOID*)&_device_linerData1);
		ExFreeMem((PVOID*)&_host_linerData2);
		ExFreeMem((PVOID*)&_host_linerData1);
		printf("�豸GPU�ڴ�������ָ��Ϊ�գ��Ѳ���.\n");
		*ret = NULL;
		exit(0xFFFF);
	}
	hipDeviceReset();
	return;
}

void ExFreeRgbTensor(
	IN_CLEAR_OUT PRGB_TENSOR* tensor
)
{
	if (*tensor == NULL)
	{
		printf("�Ѿ�Ϊ��.\n");
	}
	forss(
		(*tensor)->depth, (*tensor)->height,
		ExFreeMem((PVOID*)&(*tensor)->RGBtensor[j][i]);,
	);
	fors(
		(*tensor)->depth,
		ExFreeMem((PVOID*)&(*tensor)->RGBtensor[j]);
	);
	ExFreeMem((PVOID*)&(*tensor)->RGBtensor);
	ExFreeMem((PVOID*)&(*tensor));
	return;
}

void playRgbTensor(
	IN_OUT PRGB_TENSOR* rgbTensor
)
{
	//��ǰ��rgbTensor��һ�������������δ�ͷ��ڴ�.
	SIZE_T dimensions[MAX_DIMENSION_SIZE] = { (*rgbTensor)->height, (*rgbTensor)->width, 0 };

	return;
}

void transRgbTensor2ImgOnDisk(
	IN_CLEAR_OUT PRGB_TENSOR* rgbTensor,
	IN CONST WCHAR* imgName
)
{
	PVOID img = malloc((*rgbTensor)->width * (*rgbTensor)->height * ((*rgbTensor)->depth + 1) + 0x36 + 0x36);
	*(USHORT*)((ULONG64)img) = 0x4D42;
	*(ULONG*)((ULONG64)img + 2) = (ULONG)((*rgbTensor)->width * (*rgbTensor)->height * ((*rgbTensor)->depth + 1) + 0x36);
	*(ULONG*)((ULONG64)img + 6) = 0xCDCDCDCD;
	*(ULONG*)((ULONG64)img + 10) = 0x00000036;
	*(ULONG*)((ULONG64)img + 14) = 0x00000028;
	*(ULONG*)((ULONG64)img + 18) = (ULONG)((*rgbTensor)->width);
	*(ULONG*)((ULONG64)img + 22) = (ULONG)((*rgbTensor)->height);
	*(USHORT*)((ULONG64)img + 26) = 0x0001;
	*(USHORT*)((ULONG64)img + 28) = 0x0020;
	*(ULONG*)((ULONG64)img + 30) = 0x00000000;
	*(ULONG*)((ULONG64)img + 34) = (ULONG)((*rgbTensor)->width * (*rgbTensor)->height * ((*rgbTensor)->depth + 1));
	RtlZeroMemory((PVOID)((ULONG64)img + 38), 16);
	size_t k = (*rgbTensor)->width * (*rgbTensor)->height * ((*rgbTensor)->depth + 1) - 4;
	for (size_t j = 0; j < ((*rgbTensor)->height); j++)
	{
		for (size_t i = 0; i < ((*rgbTensor)->width); i++)
		{
			*(UCHAR*)((ULONG64)img + 54 + k + 0) = (*rgbTensor)->RGBtensor[2][j][((*rgbTensor)->width) - i - 1];
			*(UCHAR*)((ULONG64)img + 54 + k + 1) = (*rgbTensor)->RGBtensor[1][j][((*rgbTensor)->width) - i - 1];
			*(UCHAR*)((ULONG64)img + 54 + k + 2) = (*rgbTensor)->RGBtensor[0][j][((*rgbTensor)->width) - i - 1];
			*(UCHAR*)((ULONG64)img + 54 + k + 3) = 0xFF;
			k -= 4;
		}
	}
	*(ULONG*)((ULONG64)img + 54 + (ULONG64)((*rgbTensor)->width * (*rgbTensor)->height * ((*rgbTensor)->depth + 1))) = 0xFDFDFDFD;
	RtlZeroMemory((PVOID)((ULONG64)img + 54 + (ULONG64)((*rgbTensor)->width * (*rgbTensor)->height * ((*rgbTensor)->depth + 1)) + 4), 0x32);
	HANDLE file = NULL;
	CONST WCHAR* path = (CONST WCHAR*)imgName;
	file = CreateFileW(
		path,
		GENERIC_WRITE,
		0,
		NULL,
		CREATE_ALWAYS,
		FILE_ATTRIBUTE_NORMAL,
		NULL
	);
	WriteFile(file, img, (*rgbTensor)->width * (*rgbTensor)->height * ((*rgbTensor)->depth + 1) + 0x36 + 0x36, NULL, NULL);
	CloseHandle(file);
	ExFreeRgbTensor(rgbTensor);
	ExFreeMem((PVOID*)&img);
	return;
}